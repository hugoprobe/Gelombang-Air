#include "hip/hip_runtime.h"
/*Template was modified from NVIDIA Cuda Sample*/

#ifdef _MSC_VER
#define _CRT_SECURE_NO_WARNINGS
#endif

#include "header/header.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include "user defined lib/vbo.h"
#include <timer.h>
#include <numeric>
#include <iostream>

#define REFRESH_DELAY     10 //ms

const char *Judul
	= "SIMULASI GELOMBANG AIR";

// Ukuran jendela layar
unsigned int window_width  = 2000;
unsigned int window_height = 1000;

// kontrol pada mouse
int mouse_old_x, mouse_old_y;
int mouse_buttons = 0;
float rotate_x = 0.0, rotate_y = 0.0;
float translate_z = -3.0;
float translate_y = 0.0;
float translate_x = 0.0;
float translate_zz=0.0;

//variabel untuk keperluan analisis
//dan pencatatan waktu komputasi
parameter globalParam;
dataAnalysis Analis;

//Data-data komputasi dan visualisasi
GLuint shaderProg;
vbo VBO;
dataWaveCompute dataHost, dataDev;

//Variabel alokasi GPU
gpuThread GPUThread;
int STATUS=PAUSE;

void mainFlow()
{

	InitMesh(&dataHost.Mesh, "test.txt");

	InitDataComputation(&dataHost,globalParam.dt);

	if(globalParam.device==DEVICE_GPU)
	{
		GPUSimulationAllocationMemory(&dataDev, dataHost);
		threadAllocation(globalParam.threadPerBlock,
						&GPUThread,dataDev);
	}

    initVBO(&VBO,dataHost.Mesh);
    glutMainLoop();
}
void display()
{

    if(STATUS==RUNNING)
    {
    	Analis.frame++;
    	if(globalParam.device==DEVICE_GPU)
    	{
    		gpuTimingStartRec(&Analis.gCoreTime);
    		computeGPU(globalParam.iteration,GPUThread,&dataDev);
    		gpuTimingStopRec(&Analis.gCoreTime, Analis.gCoreTime.elapsed_time);
    		Analis.simulationTime+=dataDev.dt*globalParam.iteration;

    	}else
    	{
    		cpuTimingStartRec(&Analis.cCoreTime);
    		computeCPU(globalParam.iteration, &dataHost);
    		cpuTimingStopRec(&Analis.cCoreTime, Analis.cCoreTime.elapsed_time);
    		Analis.simulationTime+=dataHost.dt*globalParam.iteration;
    	}

    }

    if(!(globalParam.style==STYLE_NOVISUALIZATION))
    {

		computeVisualization(VBO,globalParam,GPUThread,&dataDev,dataHost);

//kontrol layar openGL
		glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
		glMatrixMode(GL_MODELVIEW);
		glLoadIdentity();
		glTranslatef(0.0, 0.0, translate_z);//-3.25);
		glRotatef(rotate_x+5, 1.0, 0.0, 0.0);
		glRotatef(rotate_y -5, 0.0, 1.0, 0.0);
		glTranslatef(-0.5 + translate_x, -0.5, 0.75+translate_y);

		renderingVBO(shaderProg, VBO,dataDev,globalParam);
    }

    if(STATUS==RUNNING)
		if(timesUp(globalParam.endtime,Analis.simulationTime) ||
				frameOver(globalParam.targetFrame,Analis.frame))
		  {
			cpuTimingStopRec(&Analis.cVisualTime,Analis.cVisualTime.elapsed_time);
			STATUS=PAUSE;
		  }

    glutSwapBuffers();


}


int attachShader(GLuint prg, GLenum type, const char *name)
{
    GLuint shader;
    FILE *fp;
    int size, compiled;
    char *src;

    fp = fopen(name, "rb");

    if (!fp)
    {
        return 0;
    }

    fseek(fp, 0, SEEK_END);
    size = ftell(fp);
    src = (char *)malloc(size);

    fseek(fp, 0, SEEK_SET);
    fread(src, sizeof(char), size, fp);
    fclose(fp);

    shader = glCreateShader(type);
    glShaderSource(shader, 1, (const char **)&src,
    		(const GLint *)&size);
    glCompileShader(shader);
    glGetShaderiv(shader, GL_COMPILE_STATUS,
    		(GLint *)&compiled);

    if (!compiled)
    {
        char log[2048];
        int len;

        glGetShaderInfoLog(shader, 2048,
        		(GLsizei *)&len, log);
        printf("Info log: %s\n", log);
        glDeleteShader(shader);
        return 0;
    }

    free(src);

    glAttachShader(prg, shader);
    glDeleteShader(shader);

    return 1;
}

GLuint loadGLSLProgram(const char *vertFileName,
		const char *fragFileName)
{
    GLint linked;
    GLuint program;

    program = glCreateProgram();

    if (!attachShader(program, GL_VERTEX_SHADER, vertFileName))
    {
        glDeleteProgram(program);
        fprintf(stderr, "kesalahan attach vertek %s\n", vertFileName);
        return 0;
    }

    if (!attachShader(program, GL_FRAGMENT_SHADER, fragFileName))
    {
        glDeleteProgram(program);
        fprintf(stderr, "Ckesalahan attach fragment %s\n",
        		fragFileName);

        return 0;
    }

    glLinkProgram(program);
    glGetProgramiv(program, GL_LINK_STATUS, &linked);

    if (!linked)
    {
        glDeleteProgram(program);
        char temp[256];
        glGetProgramInfoLog(program, 256, 0, temp);
        fprintf(stderr, "Failed to link program: %s\n", temp);
        return 0;
    }

    return program;
}


double cpuSecond(){
	struct timeval tp;
	gettimeofday(&tp,NULL);
	return ((double)tp.tv_sec
			+ (double)tp.tv_usec*1.e-6);
}

void gpuTimingStartRec(gpuTiming *GPUTiming)
{

		hipEventCreate( &GPUTiming->start) ;
		 hipEventCreate( &GPUTiming->stop) ;
		 hipEventRecord( GPUTiming->start, 0 ) ;
}
void gpuTimingStopRec(gpuTiming *GPUTiming,
		float prevElapsed)
{
		hipEventRecord( GPUTiming->stop, 0 ) ;
		hipEventSynchronize( GPUTiming->stop ) ;
		hipEventElapsedTime( &GPUTiming->elapsed_time,
				GPUTiming->start, GPUTiming->stop ) ;
		hipEventDestroy( GPUTiming->start ) ;
		hipEventDestroy( GPUTiming->stop ) ;
		GPUTiming->elapsed_time/=1000;
		GPUTiming->elapsed_time+=prevElapsed;

}

void cpuTimingStartRec(cpuTiming *CPUTiming)
{
	CPUTiming->start=cpuSecond();

}
void cpuTimingStopRec(cpuTiming *CPUTiming,float prevElapsed)
{
	CPUTiming->stop=cpuSecond();
	CPUTiming->elapsed_time=CPUTiming->stop-CPUTiming->start
			+prevElapsed;
}




void initParam(parameter *Param)
{
	Param->device=DEVICE_GPU;
	Param->iteration=20;
	Param->style=STYLE_SURFACE;
	Param->endtime=100;
	Param->threadPerBlock=128;
	Param->targetFrame=-1;
	Param->dt=0.00075;

}

void initAnalis(dataAnalysis *Analis)
{
	Analis->frame=0;
	Analis->cVisualTime.elapsed_time=0;
	Analis->cCoreTime.elapsed_time=0;
	Analis->gCoreTime.elapsed_time=0;

	Analis->cVisualTime.stop=0;
	Analis->cCoreTime.stop=0;
	Analis->gCoreTime.stop=0;

	Analis->cVisualTime.start=0;
	Analis->cCoreTime.start=0;
	Analis->gCoreTime.start=0;

	Analis->simulationTime=0;
}

void showAnalis(parameter P, dataAnalysis A)
{
	printf("\n\n......Data Analisis.....\n");
	printf("\nThreadPerBlock\t\t: %d",P.threadPerBlock);
	printf("\nIterationPerFrame\t: %d",P.iteration);
	printf("\nFrame\t\t\t: %d",A.frame);
	printf("\nTotal Iteration\t\t: %d",
			P.iteration*(A.frame));
	printf("\nElapsed Time CPU\t: %0.3lf",
			A.cCoreTime.elapsed_time);
	printf("\nElapsed Time GPU\t: %0.3lf",
			A.gCoreTime.elapsed_time);
	if(P.style==STYLE_NOVISUALIZATION)
		A.cVisualTime.elapsed_time=0;
	printf("\nVisualization Time\t: %0.3lf",
			A.cVisualTime.elapsed_time);
	printf("\nSimulation Real Time\t: %0.3lf",
			A.simulationTime);
	printf("\n");
}



int timesUp( double endTime,double elapsedSimulation)
{
	return (elapsedSimulation>=endTime && endTime>0);

}

int frameOver( int targetFrame,int frame)
{	glVertex3f(1.02, 0.0,0.37);
	return (frame>=targetFrame && targetFrame>0);
}



void timerEvent(int value)
{
    if (glutGetWindow())
    {
        glutPostRedisplay();
        glutTimerFunc(REFRESH_DELAY, timerEvent,0);
    }
}

void cleanup()
{

  	 deleteVBO2(&VBO.posVertexBuffer);
  	 deleteVBO2(&VBO.VertexNormalBuffer);
  	 deleteVBO2(&VBO.signBuffer);

  	 GPUSimulationFreeMemory(&dataDev);
     showAnalis(globalParam,Analis);
     glutDestroyWindow(glutGetWindow());

     hipDeviceReset();
}



void keyboard(unsigned char key, int /*x*/, int /*y*/)
{
    switch (key)
    {
    	case 13:
    		if(STATUS==PAUSE)
    		{
    			STATUS=RUNNING;
    			cpuTimingStartRec(&Analis.cVisualTime);
    		}else
    		{
    			STATUS=PAUSE;
    			cpuTimingStopRec(&Analis.cVisualTime,
    					Analis.cVisualTime.elapsed_time);
    		}
    			break;
        case (27) :
        		if(STATUS==RUNNING)
        				cpuTimingStopRec(&Analis.cVisualTime,
        						Analis.cVisualTime.elapsed_time);
            #if defined(__APPLE__) || defined(MACOSX)
                exit(EXIT_SUCCESS);
            #else


                glutLeaveMainLoop();
                return;
            #endif
    }
}

////////////////////////////////////////////////////////////////////////////////
//! Mouse event handlers
////////////////////////////////////////////////////////////////////////////////
void mouse(int button, int state, int x, int y)
{
	if (state == GLUT_DOWN && button==GLUT_MIDDLE_BUTTON)
	{
		mouse_buttons=2;

	}
	else if (state == GLUT_DOWN)
    {
        mouse_buttons |= 1<<button;
    }
    else if (state == GLUT_UP)
    {
        mouse_buttons = 0;
    }


    mouse_old_x = x;
    mouse_old_y = y;
}

void motion(int x, int y)
{
    float dx, dy;
    dx = (float)(x - mouse_old_x);
    dy = (float)(y - mouse_old_y);
    if (mouse_buttons & 1)
    {
        rotate_x += dy * 0.2f;
        rotate_y += dx * 0.2f;
    }
    else if (mouse_buttons & 4)
    {
        translate_z += dy * 0.01f;
    }
    else if (mouse_buttons ==2)
    {
    	translate_y += dy*0.01f;
    	translate_x += dx*0.01f;

    }
    mouse_old_x = x;
    mouse_old_y = y;
}


int main(int argc, char **argv)
{
    char *ref_string= NULL;

    initParam(&globalParam);
    initAnalis(&Analis);

#if defined(__linux__)
    setenv ("DISPLAY", ":0", 0);
#endif

    printf("%s starting...\n", Judul);

    if (argc > 1)
    {


        if (checkCmdLineFlag(argc, (const char **)argv, "frame"))
        {

            getCmdLineArgumentString(argc, (const char **)argv,
            		"frame", (char **)&ref_string);
            sscanf(ref_string,"%d",&globalParam.targetFrame);

        }

        if (checkCmdLineFlag(argc, (const char **)argv, "file"))
        {

            getCmdLineArgumentString(argc, (const char **)argv,
            		"file", (char **)&ref_string);
            printf("\nNama File : %s", ref_string);
            globalParam.sourceFile=ref_string;
        }
        if (checkCmdLineFlag(argc, (const char **)argv, "thread"))
		{

			getCmdLineArgumentString(argc, (const char **)argv,
					"thread", (char **)&ref_string);
			printf("\nNAlokasi Thread : %s", ref_string);
			sscanf(ref_string,"%d",&globalParam.threadPerBlock);
			printf("\nintThread : %d", globalParam.threadPerBlock);


		}
        if (checkCmdLineFlag(argc, (const char **)argv, "endtime"))
		{

			getCmdLineArgumentString(argc, (const char **)argv,
					"endtime", (char **)&ref_string);
			printf("\nEnd Time : %s", ref_string);
			sscanf(ref_string,"%lf",&globalParam.endtime);

		}
        if (checkCmdLineFlag(argc, (const char **)argv, "dt"))
		{

			getCmdLineArgumentString(argc, (const char **)argv,
					"dt", (char **)&ref_string);
			printf("\nTime Step : %s", ref_string);
			sscanf(ref_string,"%lf",&globalParam.dt);

		}
        if (checkCmdLineFlag(argc, (const char **)argv, "style"))
		{

			getCmdLineArgumentString(argc, (const char **)argv,
					"style", (char **)&ref_string);
			printf("Visual Style : %s", ref_string);
			if(strcmp(ref_string,"novisualization")==0)
			{
				globalParam.style=STYLE_NOVISUALIZATION;

			}else if(strcmp(ref_string,"surface")==0)
			{
				globalParam.style=STYLE_SURFACE;

			}else if(strcmp(ref_string,"hsv")==0)
			{
				globalParam.style=STYLE_COLORHSV;
			}
			else if(strcmp(ref_string,"wire")==0)
			 {
						globalParam.style=STYLE_WIRE;
			 }
			else
			{
				int tempInt;
				sscanf(ref_string,"%d",&tempInt);
				if(tempInt<3)
				{
					globalParam.style=tempInt;
				}
			}


		}
        if (checkCmdLineFlag(argc, (const char **)argv, "device"))
	   {

		   getCmdLineArgumentString(argc, (const char **)argv,
				   "device", (char **)&ref_string);
		   printf("\nPilihan Device: %s", ref_string);
		   if(strcmp(ref_string,"cpu")==0)
		   {
			   globalParam.device=DEVICE_CPU;
		   }else if (strcmp(ref_string,"gpu")==0)
		   {
			   globalParam.device=DEVICE_GPU;
		   }
	   }
        if (checkCmdLineFlag(argc, (const char **)argv, "iteration"))
	   {

		   getCmdLineArgumentString(argc, (const char **)argv,
				   "iteration", (char **)&ref_string);
		   printf("\nIterasi : %s", ref_string);
		   int tempInt;

			sscanf(ref_string,"%d",&tempInt);
			if(tempInt>0)
			{
				globalParam.iteration=tempInt;
			}
	   }
        if (checkCmdLineFlag(argc, (const char **)argv, "out"))
	   {

		   getCmdLineArgumentString(argc, (const char **)argv,
				   "out", (char **)&ref_string);
		   printf("\nOut file : %s", ref_string);
		   if(strcmp(ref_string,"")!=0)
		   {
			   globalParam.outFile=ref_string;
		   }
	   }
    }

    if(globalParam.style==STYLE_NOVISUALIZATION)
    {
    	STATUS=RUNNING;
    	printf("\nNO Visualization");
    	printf("\nProcess...");

    }

    runTest(argc, argv, NULL);//ref_string

}


bool initGL(int *argc, char **argv)
{
    glutInit(argc, argv);
    glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
    glutInitWindowSize(500, 500);
    glutCreateWindow("Cuda GL Interop (VBO)");
    glutFullScreen();
    glutDisplayFunc(display);
    glutKeyboardFunc(keyboard);
    glutMotionFunc(motion);
    glutTimerFunc(REFRESH_DELAY, timerEvent,0);

    char* vertShaderPath = sdkFindFilePath("test.vert", argv[0]);
    char* fragShaderPath = sdkFindFilePath("test.frag", argv[0]);
    // initialize necessary OpenGL extensions
    glewInit();

    if (! glewIsSupported("GL_VERSION_2_0 "))
    {
        fprintf(stderr, "ERROR: Support for necessary OpenGL extensions missing.");
        fflush(stderr);
        return false;
    }

    // default initialization
    glClearColor(0.0, 0.0, 0.0, 1.0);
    glDisable(GL_DEPTH_TEST);
   // viewport
    glViewport(0, 0, window_width, window_height);

    // projection
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    gluPerspective(60.0, (GLfloat)window_width / (GLfloat) window_height, 0.1, 10.0);
    shaderProg = loadGLSLProgram(vertShaderPath, fragShaderPath);
     SDK_CHECK_ERROR_GL();

    return true;
}

bool runTest(int argc, char **argv, char *ref_file)
{


    // command line mode only
    if (ref_file != NULL)
    {   printf("\nTEST");
        int devID = findCudaDevice(argc, (const char **)argv);

        hipDeviceReset();
    }
    else		glVertex3f(1.02, 0.0,0.37);
    {

        if (false == initGL(&argc, argv))
        {
            return false;
        }

        if (checkCmdLineFlag(argc, (const char **)argv, "device"))
        {
            if (gpuGLDeviceInit(argc, (const char **)argv) == -1)
            {

                return false;
            }
        }
        else
        {

            cudaGLSetGLDevice(gpuGetMaxGflopsDeviceId());
        }
       // register callbacks
        glutDisplayFunc(display);
        glutKeyboardFunc(keyboard);
        glutMouseFunc(mouse);
        glutMotionFunc(motion);
#if defined (__APPLE__) || defined(MACOSX)
        atexit(cleanup);
#else
        glutCloseFunc(cleanup);
#endif

    			char Title[BUFSIZ];
    			sprintf(Title, "Mesh Dam Break/Mesh2D_14362.neu");

                mainFlow();




    }

    return true;
}


#ifdef _WIN32
#ifndef FOPEN
#define FOPEN(fHandle,filename,mode) fopen_s(&fHandle, filename, mode)
#endif
#else
#ifndef FOPEN
#define FOPEN(fHandle,filename,mode) (fHandle = fopen(filename, mode))
#endif
#endif

void sdkDumpBin2(void *data, unsigned int bytes, const char *filename)
{
    printf("sdkDumpBin: <%s>\n", filename);
    FILE *fp;
    FOPEN(fp, filename, "wb");
    fwrite(data, bytes, 1, fp);
    fflush(fp);
    fclose(fp);
}

