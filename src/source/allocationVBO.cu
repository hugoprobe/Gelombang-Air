#include "hip/hip_runtime.h"
#include "../../header/header.h"
#include "../../user defined lib/vbo.h"



void createMeshIndexBuffer(GLuint *id, mesh Mesh)
		//int *EtoV, int Nelems, int NNodes)
{
    int i, size = (Mesh.NCells+Mesh.Wall.count*6)*3*sizeof(GLuint);
    int endPosVertex=Mesh.NNodes;
    // create index buffer
    glGenBuffersARB(1, id);
    glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, *id);
    glBufferDataARB(GL_ELEMENT_ARRAY_BUFFER, size, 0, GL_STATIC_DRAW);

    // fill with indices for rendering mesh as triangle strips
    GLuint *indices = (GLuint *) glMapBuffer(GL_ELEMENT_ARRAY_BUFFER, GL_WRITE_ONLY);

    if (!indices)
    {
        return;
    }

    for (i=0; i<Mesh.NCells*3; i++)
    {
    	*indices++=Mesh.EtoV[i];
    	//printf("\nEtoV %d %d %d", i, i%3,Mesh.EtoV[i]);
    }
    for(i=0; i<Mesh.Wall.count*6;i++)
    {
    	*indices++=endPosVertex++;
    }


    glUnmapBuffer(GL_ELEMENT_ARRAY_BUFFER);
    glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, 0);

}

// create fixed vertex buffer to store mesh vertices
void createMeshPositionVBO(GLuint *id, struct hipGraphicsResource **vertex_pos, mesh Mesh)
		//double * VertX, double *VertY, int Nnodes)
{
    createVBO2(id, (Mesh.NNodes+Mesh.Wall.count*6)*4*sizeof(float));


    glBindBuffer(GL_ARRAY_BUFFER, *id);
    float *pos = (float *) glMapBuffer(GL_ARRAY_BUFFER, GL_WRITE_ONLY);

    if (!pos)
    {
        return;
    }

    for (int idx=0; idx<Mesh.NNodes; idx++)
    {

     //       float u = x / (float)(w-1);
      //      float v = y / (float)(h-1);
            *pos++ = Mesh.VertX[idx];
            *pos++ = 0.0f;
            *pos++ = Mesh.VertY[idx];
            *pos++ = 1.0f;

        //    printf("\nCoor %lf    %lf", Mesh.VertX[idx], Mesh.VertY[idx]);

    }



    for (int idx=0; idx<Mesh.Wall.count; idx++)
    {
    	float3 v0=make_float3(Mesh.Wall.Point1[idx].x,Mesh.Wall.Point1[idx].y,0);

    	float3 edge1=Mesh.Wall.Point2[idx]-Mesh.Wall.Point1[idx];
    	float3 edge2=v0-Mesh.Wall.Point1[idx];

    	float3 normal=cross(edge1,edge2);
/*
    	for(int i=0;i<6;i++)
    		normalWall[idx*6+i]=normal;
*/

            *pos++ =  Mesh.Wall.Point1[idx].x;
            *pos++ = 0.0f;
            *pos++ = Mesh.Wall.Point1[idx].y;
            *pos++=1.0;

            *pos++ =  Mesh.Wall.Point1[idx].x;
            *pos++ = Mesh.Wall.Point1[idx].z;
       //     printf("\n%f",*(pos-1));
            *pos++ = Mesh.Wall.Point1[idx].y;
            *pos++=1.0;

            *pos++ =  Mesh.Wall.Point2[idx].x;
            *pos++ = Mesh.Wall.Point2[idx].z;
        //    printf("\n%f",*(pos-1));
            *pos++ = Mesh.Wall.Point2[idx].y;
            *pos++=1.0;





            *pos++ =  Mesh.Wall.Point1[idx].x;
		    *pos++ = 0.0f;
		    *pos++ = Mesh.Wall.Point1[idx].y;
		    *pos++=1.0;

            *pos++ =  Mesh.Wall.Point2[idx].x;
            *pos++ = 0.0f;
            *pos++ = Mesh.Wall.Point2[idx].y;

            *pos++=1.0;


            *pos++ =  Mesh.Wall.Point2[idx].x;
			*pos++ = Mesh.Wall.Point2[idx].z;
		//	printf("\n%f",*(pos-1));
			*pos++ = Mesh.Wall.Point2[idx].y;
			*pos++=1.0;





    }

    glUnmapBuffer(GL_ARRAY_BUFFER);
    glBindBuffer(GL_ARRAY_BUFFER, 0);


    checkCudaErrors(hipGraphicsGLRegisterBuffer(vertex_pos, *id, cudaGraphicsMapFlagsNone));
}

void initVBO(vbo *VBO, mesh Mesh)
{
    createVBO2(&VBO->VertexNormalBuffer, (Mesh.NNodes+Mesh.Wall.count*6)*sizeof(float3));

		       glBindBuffer(GL_ARRAY_BUFFER, VBO->VertexNormalBuffer);
		             float3 *pos = (float3 *) glMapBuffer(GL_ARRAY_BUFFER, GL_WRITE_ONLY);

		             if (!pos)
		             {
		               //  return;

		             }else
		             {

		            	 pos=pos+Mesh.NNodes;

		            	 for(int ii=0;ii<Mesh.Wall.count;ii++)
		            	 {
		            		 int s=-1;
		            		 if(ii%3<3)s*=-1;
		            		 float3 v0=make_float3(Mesh.Wall.Point1[ii].x,Mesh.Wall.Point1[ii].y,0);
		            		 float3 edge1=Mesh.Wall.Point2[ii]-Mesh.Wall.Point1[ii];
							float3 edge2=v0-Mesh.Wall.Point1[ii];

							float3 normal=make_float3(0,0,0);//cross(edge2,edge1);

							for(int i=0;i<6;i++)

								*pos++=normal;


						//	make_float3
		            	 }

		             }
				 glUnmapBuffer(GL_ARRAY_BUFFER);
				 glBindBuffer(GL_ARRAY_BUFFER, 0);
				 //make_float3



		       checkCudaErrors(hipGraphicsGLRegisterBuffer(&VBO->cuda_VertexNormal_resource, VBO->VertexNormalBuffer, cudaGraphicsMapFlagsWriteDiscard));

		      createVBO2(&VBO->signBuffer, Mesh.NNodes*sizeof(float));
		      checkCudaErrors(hipGraphicsGLRegisterBuffer(&VBO->cuda_Sign_resource,VBO->signBuffer, cudaGraphicsMapFlagsWriteDiscard));

		      createMeshPositionVBO(&VBO->posVertexBuffer, &VBO->cuda_VertexPos_resource, Mesh);
	          createMeshIndexBuffer(&VBO->indexBuffer, Mesh);



}
