#include "hip/hip_runtime.h"
#include "../../header/header.h"



__global__ void kernelUpdateVertex(float4 *pos,int *VtoE, int *VtoS, double *VertX, double *VertY, double *p , int maxconn, int Nnodes)
{


	unsigned int idx=threadIdx.x + blockIdx.x * blockDim.x;

	double w;

	int i, k;
	float sum=0;
	int ie;
	int iconn=0;
	if(idx>=Nnodes) return;



	k=idx*maxconn;
	for(i=0;  i < maxconn; i++)
	{

		ie = VtoE[k+i];
		if(ie==-1)break;
		sum=sum+p[ie];
		iconn++;


	}

	w=1.0*sum/iconn;


	pos[idx]=make_float4(pos[idx].x,w,pos[idx].z, .0f);


}

__global__ void kernelNormalFace(float4 *vertexPos, float3 *normalFace, int *EtoV, int Nelems)
{

	float3 edge1,edge2;
	int i1,i2,i3;

	unsigned int idx=threadIdx.x + blockIdx.x * blockDim.x;

	if(idx>=Nelems) return;

	i1 = EtoV[idx*3 + 0];
	i2 = EtoV[idx*3 + 1];
	i3 = EtoV[idx*3 + 2];


	edge1=operator-(make_float3(vertexPos[i2]),make_float3(vertexPos[i1]));
	edge2=operator-(make_float3(vertexPos[i3]),make_float3(vertexPos[i1]));



	normalFace[idx] = cross(edge2,edge1);


}

__global__ void kernelNormalVektor(float4 *vertexPos, float3 *normalVector, float3 *normalFace, int *VtoE, int maxconn, int Nnodes)
{
	unsigned int idx=threadIdx.x + blockIdx.x * blockDim.x;
	float3 vsum=make_float3(0.0,0.0,0.0);
	int i, k=idx*maxconn;
	if(idx>=Nnodes) return;

		for(i=0;  i < maxconn; i++)
		{

			int ie = VtoE[k+i];
			if(ie==-1)break;
			operator+=(vsum, normalFace[ie]);

		}

		normalVector[idx]=operator/(vsum,i);
}


void computeVisualization(vbo VBO, parameter Param,gpuThread  GPUThread,  dataWaveCompute *dataDev, dataWaveCompute dataHost)
{
	float4 *vertexPos;
	float3 *nptr;
	size_t num_bytes;

	if(Param.device==DEVICE_CPU) GPUSimulationAllocationMemory(dataDev,dataHost);


	checkCudaErrors(hipGraphicsMapResources(1, &VBO.cuda_VertexNormal_resource, 0));
		    checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&nptr, &num_bytes, VBO.cuda_VertexNormal_resource));


		    checkCudaErrors(hipGraphicsMapResources(1, &VBO.cuda_VertexPos_resource, 0));
		    checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&vertexPos, &num_bytes,
		    		VBO.cuda_VertexPos_resource));

		kernelUpdateVertex<<<GPUThread.grid2,GPUThread.block>>>(vertexPos,dataDev->Mesh.VtoE, dataDev->Mesh.VtoS, dataDev->Mesh.VertX,
						dataDev->Mesh.VertY, dataDev->h, dataDev->Mesh.maxVtoEconn,	dataDev->Mesh.NNodes);



		kernelNormalFace<<<GPUThread.grid1,GPUThread.block>>>(vertexPos, dataDev->Mesh.NormalFaces, dataDev->Mesh.EtoV, dataDev->Mesh.NCells);

		kernelNormalVektor<<<GPUThread.grid2,GPUThread.block>>>(vertexPos, nptr, dataDev->Mesh.NormalFaces, dataDev->Mesh.VtoE,
						dataDev->Mesh.maxVtoEconn, dataDev->Mesh.NNodes);

		checkCudaErrors(hipGraphicsUnmapResources(1, &VBO.cuda_VertexNormal_resource, 0));


		 checkCudaErrors(hipGraphicsUnmapResources(1, &VBO.cuda_VertexPos_resource, 0));
	 if(Param.device==DEVICE_CPU)  GPUSimulationFreeMemory(dataDev);


}


