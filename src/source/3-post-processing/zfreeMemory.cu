#include "../../header/header.h"


double GPUSimulationFreeMemory(dataWaveCompute *dataDev)
{



				float gpu_time=0.0;
				hipEvent_t start,stop;
					hipEventCreate( &start ) ;
					 hipEventCreate( &stop ) ;
					 hipEventRecord( start, 0 ) ;


		hipFree(dataDev->h);
		hipFree(dataDev->u);
		hipFree(dataDev->v);
		hipFree(dataDev->hu);
		hipFree(dataDev->hv);
		hipFree(dataDev->z);


		hipFree(dataDev->hnew);
		hipFree(dataDev->unew);
		hipFree(dataDev->vnew);
		hipFree(dataDev->hunew);
		hipFree(dataDev->hvnew);

		hipFree(dataDev->Mesh.L);
		hipFree(dataDev->Mesh.EtoE);
		hipFree(dataDev->Mesh.EtoV);
		hipFree(dataDev->Mesh.VtoE);
		hipFree(dataDev->Mesh.VtoS);
		hipFree(dataDev->Mesh.NormalFaces);
		hipFree(dataDev->Mesh.nx);
		hipFree(dataDev->Mesh.ny);
		hipFree(dataDev->Mesh.VertX);
		hipFree(dataDev->Mesh.VertY);
		hipFree(dataDev->Mesh.AREA);

		 hipEventRecord( stop, 0 ) ;
		 hipEventSynchronize( stop ) ;
		hipEventElapsedTime( &gpu_time,	start, stop ) ;
		hipEventDestroy( start ) ;
		hipEventDestroy( stop ) ;

		hipDeviceSynchronize();


		return (double)gpu_time/1000.0;
}
