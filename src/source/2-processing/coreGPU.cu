#include "hip/hip_runtime.h"
#include "../../header/header.h"

inline __host__ __device__ int isIn(int elem)
{
	unsigned int indicesLd[12]={21818,21822,21826,21830,21834,21838,21842,21846,21850,21854,21858,21862};
	unsigned int indicesRd[12]={24412,24416,24420,24424,244428,24432,24436,24440,24444,24448,24452,24456};

	for(int i =0;i<12;i++)
	{
		if(elem==indicesLd[i] || elem==indicesRd[i])
		{
			return 1;

		}

	}
	return 0;
}


inline __host__ __device__ double calculate_a(double *a_plus, double *a_min, double uj, double uk, double hj, double hk)
{
	*a_plus=max(uj+sqrt(g*hj),uk+sqrt(g*hk));
	*a_plus=max(*a_plus,0.0);
	*a_min=min(uj-sqrt(g*hj),uk-sqrt(g*hk));
	*a_min=min(*a_min,0.0);
	return 1;
}

inline __host__ __device__ double calculate_H(double HUj, double HUk, double a_plus, double a_min, double Uj,double Uk)
{
	double da=a_plus-a_min;
	if(da<10e-8) return 0.5*(HUj-HUk);
	return ((a_plus*HUj - a_min*HUk)/da + a_plus*a_min/da*(Uk-Uj));

}

inline __host__ __device__ double calculate_Unew(double Uold, double EFluxCrossL, double dt, double A)
{
	return (Uold-dt*EFluxCrossL/A);
}

inline __host__ __device__ int isReflectif(int nb, int k)
{
	return (nb==k || isIn(nb));
}


__global__ void kernelcopy(
		double *h1, double *u1, double *v1, double *hu1, double *hv1,
		double *h, double *u, double *v, double *hu, double *hv, int Nelems)
{
	unsigned int idx=threadIdx.x + blockIdx.x * blockDim.x;

	if(idx>=Nelems) return;

	h[idx]=h1[idx];
	hu[idx]=hu1[idx];
	hv[idx]=hv1[idx];

	u[idx]=hu[idx]*1.0/(h[idx]*1.0);
	v[idx]=hv[idx]*1.0/(h[idx]*1.0);


}


__global__ void kernelFluxCalculation( int *EtoV,
		double *h1, double *u1, double *v1, double *hu1, double *hv1,
		double *h, double *u, double *v, double *hu, double *hv,

		double *L, int *EtoE, double *normx, double *normy, double *AREA, double dt, int Nelems)

{
	unsigned int idx=threadIdx.x + blockIdx.x*blockDim.x;
	if(idx>=Nelems) return;
	//if(h1[idx]>0.01 && h1[idx]<0.3)printf("\n Test %d %lf  %lf  %lf",idx, v[idx],h1[idx],hu[idx]);
	double hj, hk, uj, uk, vj,vk, huj, huk, hvj, hvk;
	double a_plus, a_min;
	double HUj, HUk, nx, ny, l, A;
	double Flux_h=0.0, Flux_hu=0.0, Flux_hv=0.0;


	//fill common elements
	hj=h[idx];
	uj=u[idx];
	vj=v[idx];
	huj=hu[idx];
	hvj=hv[idx];
	A=AREA[idx];
	uj=huj/hj;
	vj=hvj/hj;




	//In every triangles, visit its neighbor, using this loop
	for(int n=0; n<3;n++)
	{
			//using k to get the mapping index of neighbor(nb) element on EtoE
			//save in nb
			int k = idx*3+ n;
			int nb= EtoE[k];



			//fill neighbor elements

			nx=normx[k];

			ny=normy[k];
			l=L[k];

			hk=h[nb];
							uk=u[nb];
							vk=v[nb];
							huk=hu[nb];
							hvk=hv[nb];
							uk=huk/hk;
							vk=hvk/hk;

			if(nb==idx)
			{

				hk=hj;
				uk=-uj;
				vk=-vj;
				huk=-huj;
				hvk=-hvj;

			}



			//Calculate wave speed
		//	calculate_a(&a_plus, &a_min,sqrt(uj*uj+vj*vj), sqrt(uk*uk+vk*vk),hj,hk);

		//	calculate_a(&a_plus, &a_min,uj, uk,hj,hk);
			calculate_a(&a_plus, &a_min,nx*uj +ny*vj, nx*uk+ny*vk,hj,hk);

			//Calculate Height Flux
			HUj= huj*nx + hvj*ny;
			HUk= huk*nx + hvk*ny;
			Flux_h+=l*calculate_H(HUj, HUk, a_plus, a_min, hj, hk);

			//Calculate X momentum Flux
			HUj = (huj*uj + 0.5*g*hj*hj)*nx;
			HUj+= (huj*vj)*ny;
			HUk = (huk*uk + 0.5*g*hk*hk)*nx;
			HUk+= (huk*vk)*ny;
		//	Flux_hu+=l*calculate_H(HUj, HUk, a_plus, a_min, huj, isReflectif(nb,idx)?-huk:huk);
			Flux_hu+=l*calculate_H(HUj, HUk, a_plus, a_min, huj, huk);

			//Calculate Y momentum Flux
			HUj = (hvj*uj)*nx;
			HUj+= (hvj*vj + 0.5*g*hj*hj)*ny;
			HUk = (hvk*uk)*nx;
			HUk+= (hvk*vk + 0.5*g*hk*hk)*ny;
		//	Flux_hv+=l*calculate_H(HUj, HUk, a_plus, a_min, hvj,  isReflectif(nb,idx)?-hvk:hvk);
			Flux_hv+=l*calculate_H(HUj, HUk, a_plus, a_min, hvj,  hvk);

	}



	h1[idx]=calculate_Unew(hj,Flux_h, dt,A) ;
	hu1[idx]=calculate_Unew(huj, Flux_hu,dt,A);
	hv1[idx]=calculate_Unew(hvj, Flux_hv,dt,A);



}




double computeGPU(int iteration , gpuThread GPUThread,   dataWaveCompute *dataDev)
{

	for(int i=0;i<iteration;i++)
	{

		kernelFluxCalculation<<<GPUThread.grid1,GPUThread.block>>>( dataDev->Mesh.EtoV,
				dataDev->hnew, dataDev->unew, dataDev->vnew, dataDev->hunew, dataDev->hvnew,
				dataDev->h, dataDev->u, dataDev->v, dataDev->hu, dataDev->hv,
				dataDev->Mesh.L,
				dataDev->Mesh.EtoE, dataDev->Mesh.nx, dataDev->Mesh.ny,
				dataDev->Mesh.AREA, dataDev->dt, dataDev->Mesh.NCells);

		kernelcopy<<<GPUThread.grid1,GPUThread.block>>>(dataDev->hnew,dataDev->unew, dataDev->vnew, dataDev->hunew, dataDev->hvnew,
									dataDev->h, dataDev->u, dataDev->v, dataDev->hu, dataDev->hv,dataDev->Mesh.NCells);
	}


	return 0;


}
