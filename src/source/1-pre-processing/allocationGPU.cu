#include "hip/hip_runtime.h"
#include "../../header/header.h"



double GPUSimulationAllocationMemory(dataWaveCompute *dataDev, dataWaveCompute dataHost){

		hipSetDevice(0);
		float gpu_time=0.0;
		hipEvent_t start,stop;
			hipEventCreate( &start ) ;
			 hipEventCreate( &stop ) ;
			 hipEventRecord( start, 0 ) ;


		dataDev->dt=dataHost.dt;

		dataDev->Mesh=dataHost.Mesh;

		hipMalloc((double**)&dataDev->h, dataDev->Mesh.NCells * sizeof(double));
		hipMalloc((double**)&dataDev->u, dataDev->Mesh.NCells* sizeof(double));
		hipMalloc((double**)&dataDev->v, dataDev->Mesh.NCells* sizeof(double));
		hipMalloc((double**)&dataDev->hu, dataDev->Mesh.NCells* sizeof(double));
		hipMalloc((double**)&dataDev->hv, dataDev->Mesh.NCells* sizeof(double));

		hipMalloc((double**)&dataDev->hnew, dataDev->Mesh.NCells * sizeof(double));
		hipMalloc((double**)&dataDev->unew, dataDev->Mesh.NCells* sizeof(double));
		hipMalloc((double**)&dataDev->vnew, dataDev->Mesh.NCells* sizeof(double));
		hipMalloc((double**)&dataDev->hunew, dataDev->Mesh.NCells* sizeof(double));
		hipMalloc((double**)&dataDev->hvnew, dataDev->Mesh.NCells* sizeof(double));

		hipMalloc((double**)&dataDev->Mesh.L, dataDev->Mesh.NCells *3* sizeof(double));
		hipMalloc((int**)&dataDev->Mesh.EtoE, dataDev->Mesh.NCells *3* sizeof(int));
		hipMalloc((int**)&dataDev->Mesh.EtoV, dataDev->Mesh.NCells *3* sizeof(int));
		hipMalloc((int**)&dataDev->Mesh.VtoE, dataDev->Mesh.NNodes * dataDev->Mesh.maxVtoEconn* sizeof(int));
		hipMalloc((int**)&dataDev->Mesh.VtoS, dataDev->Mesh.NNodes * dataDev->Mesh.maxVtoEconn* sizeof(int));


		hipMalloc((double**)&dataDev->Mesh.nx, dataDev->Mesh.NCells *3* sizeof(double));
		hipMalloc((double**)&dataDev->Mesh.ny, dataDev->Mesh.NCells *3* sizeof(double));
		hipMalloc((double**)&dataDev->Mesh.AREA, dataDev->Mesh.NCells* sizeof(double));
		hipMalloc((double**)&dataDev->Mesh.VertX, dataDev->Mesh.NNodes* sizeof(double));
		hipMalloc((double**)&dataDev->Mesh.VertY, dataDev->Mesh.NNodes* sizeof(double));

		hipMalloc((double**)&dataDev->Mesh.NormalFaces, dataDev->Mesh.NCells* sizeof(float3));

//		hipMalloc((double**)&dataDev->z, dataDev->Mesh.NCells* sizeof(double));



		hipMemcpy(dataDev->h, dataHost.h, dataDev->Mesh.NCells * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(dataDev->u, dataHost.u, dataDev->Mesh.NCells * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(dataDev->v, dataHost.v, dataDev->Mesh.NCells * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(dataDev->hu, dataHost.hu, dataDev->Mesh.NCells * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(dataDev->hv, dataHost.hv, dataDev->Mesh.NCells * sizeof(double), hipMemcpyHostToDevice);


		hipMemcpy(dataDev->Mesh.L, dataHost.Mesh.L, dataDev->Mesh.NCells * 3*sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(dataDev->Mesh.EtoE, dataHost.Mesh.EtoE, dataDev->Mesh.NCells * 3*sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(dataDev->Mesh.EtoV, dataHost.Mesh.EtoV, dataDev->Mesh.NCells * 3*sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(dataDev->Mesh.VtoE, dataHost.Mesh.VtoE, dataDev->Mesh.NNodes * dataDev->Mesh.maxVtoEconn*sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(dataDev->Mesh.VtoS, dataHost.Mesh.VtoS, dataDev->Mesh.NNodes * dataDev->Mesh.maxVtoEconn*sizeof(int), hipMemcpyHostToDevice);

		hipMemcpy(dataDev->Mesh.nx, dataHost.Mesh.nx, dataDev->Mesh.NCells * 3*sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(dataDev->Mesh.ny, dataHost.Mesh.ny, dataDev->Mesh.NCells * 3*sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(dataDev->Mesh.AREA, dataHost.Mesh.AREA, dataDev->Mesh.NCells * sizeof(double), hipMemcpyHostToDevice);
//		hipMemcpy(dataDev->z, dataHost.z, dataDev->Mesh.NCells * sizeof(double), hipMemcpyHostToDevice);
	//			printf("\nTest... count %doperator %lf", dataDev->Mesh.NCells, dataHost.Mesh.AREA[221]);
	//			exit(-1);

		hipMemcpy(dataDev->Mesh.VertX, dataHost.Mesh.VertX, dataDev->Mesh.NNodes * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(dataDev->Mesh.VertY, dataHost.Mesh.VertY, dataDev->Mesh.NNodes * sizeof(double), hipMemcpyHostToDevice);


		 hipEventRecord( stop, 0 ) ;
		 hipEventSynchronize( stop ) ;
		hipEventElapsedTime( &gpu_time,	start, stop ) ;
		hipEventDestroy( start ) ;
		hipEventDestroy( stop ) ;

		return (double)gpu_time/1000.0;

}


int threadAllocation(int blockAllocation, gpuThread *GPUThread, dataWaveCompute dataDev)
{
	dim3 block(blockAllocation);
	dim3 grid1 ((dataDev.Mesh.NCells+ block.x-1)/block.x);
	dim3 grid2((dataDev.Mesh.NCells+ block.x-1)/block.x);

	GPUThread->grid1=grid1;
	GPUThread->grid2=grid2;
	GPUThread->block=block;

	return 1;
}
